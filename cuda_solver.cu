#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime_api.h>
#include<helper_timer.h>
#include"move_defines.h"
#include"solver_struct.h"
#include"cuda_solver.h"

#if (N_GPU > 0)

uint16_t *CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_ORIENTATION[N_STREAM/2] = {NULL};//76.9KB
uint16_t *CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_12_FLIP[N_STREAM/2] = {NULL};//72.0KB
uint16_t *CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_4_POSITION[N_STREAM/2] = {NULL};//17.4KB
uint16_t *CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_POSITION[N_STREAM/2] = {NULL};

__constant__ uint8_t CUDA_SOLVER_MOV_TRS_TAB_LR[N_MOVES];
__constant__ uint8_t CUDA_SOLVER_MOV_TRS_TAB_FB[N_MOVES];
uint8_t *CUDA_SOLVER_DISTANCE_TABLE[N_STREAM/2] = {NULL};
uint8_t *CUDA_SOLVER_CORNER_DISTANCE_TABLE[N_STREAM/2] = {NULL};

uint16_t *CUDA_SOLVER_CORNER_ORIENTATION_SYM_to_SYM0[N_STREAM/2] = {NULL};
__constant__ uint16_t CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0[N_EDGE_FLIP * N_SYM];
__constant__ uint8_t D_INVALID_MOVE[7];


move_hist *D_MOVE_HIST[N_STREAM];
search_node_cube *D_SEARCH_NODE[N_STREAM];
uint32_t *D_PHASE2_ENTRY[N_STREAM];
int32_t *D_RESULT[N_STREAM];
hipStream_t STREAM[N_STREAM];


__device__
void d_search_node_level_move(search_node_cube_ptr cube, int32_t mov_id,
uint16_t *d_corner_8pos_mov_table,
uint16_t *d_corner_8ori_mov_table,
uint16_t *d_edge_4pos_mov_table,
uint16_t *d_edge_12flip_mov_table
){
  int32_t mv_lr,mv_fb;
  cube->corner_position       = d_corner_8pos_mov_table[cube->corner_position       * N_MOVES + mov_id];
  cube->ud_corner_orientation = d_corner_8ori_mov_table[cube->ud_corner_orientation * N_MOVES + mov_id];
  cube->ud_edge_flip          = d_edge_12flip_mov_table[cube->ud_edge_flip          * N_MOVES + mov_id];
  cube->ud_edge_ud            = d_edge_4pos_mov_table  [cube->ud_edge_ud            * N_MOVES + mov_id];
  mv_lr = CUDA_SOLVER_MOV_TRS_TAB_LR[mov_id];
  cube->lr_corner_orientation = d_corner_8ori_mov_table[cube->lr_corner_orientation * N_MOVES + mv_lr];
  cube->lr_edge_flip          = d_edge_12flip_mov_table[cube->lr_edge_flip          * N_MOVES + mv_lr];
  cube->lr_edge_ud            = d_edge_4pos_mov_table  [cube->lr_edge_ud            * N_MOVES + mv_lr];
  mv_fb = CUDA_SOLVER_MOV_TRS_TAB_FB[mov_id];
  cube->fb_corner_orientation = d_corner_8ori_mov_table[cube->fb_corner_orientation * N_MOVES + mv_fb];
  cube->fb_edge_flip          = d_edge_12flip_mov_table[cube->fb_edge_flip          * N_MOVES + mv_fb];
  cube->fb_edge_ud            = d_edge_4pos_mov_table  [cube->fb_edge_ud            * N_MOVES + mv_fb];

}
#define dist_c(i) ((d_corner_distance_table[(i)>>1] >> (((i)&1)<<2))&0x0F)

__global__
void search_tree_phase2_cuda_karnel(
  search_node_cube_ptr d_bridge_cube_node,
  move_hist_ptr move_hist,
  uint32_t *d_bridge_entry_count,
  int32_t * result,
  uint16_t *d_corner_8pos_mov_table,
  uint16_t *d_corner_8ori_mov_table,
  uint16_t *d_edge_4pos_mov_table,
  uint16_t *d_edge_12flip_mov_table,
  uint8_t *d_distance_table,
  uint8_t *d_corner_distance_table,
  uint16_t *corner_ori_sym
){
  int32_t i;
  uint32_t mh,ml;
  uint32_t mh_bk, ml_bk;
  int32_t remain_depth;
  int32_t mov;
  int32_t node_idx;
  search_node node_arr[PHASE2_SEARCH_DEPTH + 2];
  search_node curr_node;
  search_node next_node;

  i = blockIdx.x*blockDim.x + threadIdx.x;
  curr_node.cube = d_bridge_cube_node[0];

  mh = mh_bk = move_hist[i].move_hist_hi;
  ml = ml_bk = move_hist[i].move_hist_lo;

  mov = N_MOVES;

  while(mh){
    mov = (mh & 0x1F)-1;
    d_search_node_level_move(&curr_node.cube,mov,
                  d_corner_8pos_mov_table,
                  d_corner_8ori_mov_table,
                  d_edge_4pos_mov_table,
                  d_edge_12flip_mov_table
                  );

    mh >>= 5;
  }
  while(ml){
    mov = (ml & 0x1F)-1;
    d_search_node_level_move(&curr_node.cube,mov,
                  d_corner_8pos_mov_table,
                  d_corner_8ori_mov_table,
                  d_edge_4pos_mov_table,
                  d_edge_12flip_mov_table
                  );

    ml >>= 5;
  }

  curr_node.mov = mov;
  node_arr[0] = curr_node;
  node_arr[1].mov = -1;

  node_idx = 0;
  remain_depth = PHASE2_SEARCH_DEPTH;

  while(node_idx >=0 && result[0] == -1){
    curr_node = node_arr[node_idx];
    next_node = node_arr[node_idx + 1];
    if(remain_depth == 0){
      if(result[0] == -1){
        int32_t ii,jj = 0;
        if(atomicCAS(&result[0],-1,0)==-1){
          mh = mh_bk;
          ml = ml_bk;
          while(mh){
            result[jj++] = (mh & 0x1F)-1;
            mh >>= 5;
          }
          while(ml){
            result[jj++] = (ml & 0x1F)-1;
            ml >>= 5;
          }
          for(ii = 0;ii<PHASE2_SEARCH_DEPTH;ii++){
            result[jj+ii] = node_arr[ii+1].mov;
          }
        }
        break;
      }
      node_idx--;
      remain_depth ++;
    }else{
      for(mov = next_node.mov + 1; mov < N_MOVES; mov ++){
        int32_t mv_lr, mv_fb;
        int32_t sym;
        int32_t c_ori, c_ori_sym, c_ori_ud, c_pos;
        uint32_t flip, flip_sym, class_;
        int32_t edge_4pos;
        uint32_t index1, index2;
        if((D_INVALID_MOVE[(curr_node.mov * 0x16)>>6]>>((mov * 0x16)>>6))&1){
          continue;
        }
//================================================================
        c_ori = d_corner_8ori_mov_table[curr_node.cube.ud_corner_orientation * N_MOVES + mov];
        c_ori_ud = c_ori;
        next_node.cube.ud_corner_orientation = c_ori;
        flip = d_edge_12flip_mov_table[curr_node.cube.ud_edge_flip          * N_MOVES + mov];
        next_node.cube.ud_edge_flip = flip;
        edge_4pos = d_edge_4pos_mov_table[curr_node.cube.ud_edge_ud            * N_MOVES + mov];
        next_node.cube.ud_edge_ud = edge_4pos;
        sym = edge_4pos & 0x07;
        class_ = edge_4pos >> 3;
        c_ori_sym = corner_ori_sym[c_ori * N_SYM + sym];
        flip_sym = CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0[flip * N_SYM + sym];
        index1 = class_* (N_EDGE_FLIP/2) * N_CORNER_ORI + ((flip_sym * N_CORNER_ORI + c_ori_sym)>>1);
        index2 = ((flip_sym + c_ori_sym)&1)<<2;
        if(remain_depth - 1 < ((d_distance_table[index1] >> index2)&0x0F)){
          continue;
        }
        mv_lr = CUDA_SOLVER_MOV_TRS_TAB_LR[mov];
        c_ori = d_corner_8ori_mov_table[curr_node.cube.lr_corner_orientation * N_MOVES + mv_lr];
        next_node.cube.lr_corner_orientation = c_ori;
        flip = d_edge_12flip_mov_table[curr_node.cube.lr_edge_flip          * N_MOVES + mv_lr];
        next_node.cube.lr_edge_flip = flip;
        edge_4pos = d_edge_4pos_mov_table[curr_node.cube.lr_edge_ud            * N_MOVES + mv_lr];
        next_node.cube.lr_edge_ud = edge_4pos;
        sym = edge_4pos & 0x07;
        class_ = edge_4pos >> 3;
        c_ori_sym = corner_ori_sym[c_ori * N_SYM + sym];
        flip_sym = CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0[flip * N_SYM + sym];
        index1 = class_* (N_EDGE_FLIP/2) * N_CORNER_ORI + ((flip_sym * N_CORNER_ORI + c_ori_sym)>>1);
        index2 = ((flip_sym + c_ori_sym)&1)<<2;
        if(remain_depth - 1 < ((d_distance_table[index1] >> index2)&0x0F)){
          continue;
        }
        mv_fb = CUDA_SOLVER_MOV_TRS_TAB_FB[mov];
        c_ori = d_corner_8ori_mov_table[curr_node.cube.fb_corner_orientation * N_MOVES + mv_fb];
        next_node.cube.fb_corner_orientation = c_ori;
        flip = d_edge_12flip_mov_table[curr_node.cube.fb_edge_flip          * N_MOVES + mv_fb];
        next_node.cube.fb_edge_flip = flip;
        edge_4pos = d_edge_4pos_mov_table[curr_node.cube.fb_edge_ud            * N_MOVES + mv_fb];
        next_node.cube.fb_edge_ud = edge_4pos;
        sym = edge_4pos & 0x07;
        class_ = edge_4pos >> 3;
        c_ori_sym = corner_ori_sym[c_ori * N_SYM + sym];
        flip_sym = CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0[flip * N_SYM + sym];
        index1 = class_* (N_EDGE_FLIP/2) * N_CORNER_ORI + ((flip_sym * N_CORNER_ORI + c_ori_sym)>>1);
        index2 = ((flip_sym + c_ori_sym)&1)<<2;
        if(remain_depth - 1 < ((d_distance_table[index1] >> index2)&0x0F)){
          continue;
        }

        c_pos = d_corner_8pos_mov_table[curr_node.cube.corner_position * N_MOVES + mov];
        if (remain_depth - 1 < dist_c(c_pos * N_CORNER_ORI + c_ori_ud)){
          continue;
        }
        next_node.cube.corner_position = c_pos;
//================================================================
        next_node.mov = mov;
        node_arr[node_idx + 1] = next_node;
        break;
      }
      if(mov == N_MOVES){
        node_idx--;
        remain_depth ++;
      }else{
        node_idx++;
        remain_depth --;
        node_arr[node_idx + 1].mov = -1;
      }
    }
  }
}

static void HandleError(hipError_t err, const char *file, int line)
{
    if(err!=hipSuccess){
    printf("%s in %s file at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

void search_tree_phase2_cuda_async(phase2_chunk_ptr chunk, int32_t stream_id){
  int32_t i;
  int32_t result_init = -1;
  int32_t entry = chunk->entry;
  int32_t device = stream_id/2;
  hipSetDevice(device);

  if(entry%BLOCK_DIM){
    for(i = entry;i<(entry+BLOCK_DIM-1)/BLOCK_DIM*BLOCK_DIM;i++){
      chunk->move_hist[i] = chunk->move_hist[entry - 1];
    }
    entry = (entry+BLOCK_DIM-1)/BLOCK_DIM*BLOCK_DIM;
  }

  HANDLE_ERROR(hipMemcpyAsync(D_MOVE_HIST[stream_id],chunk->move_hist,sizeof(move_hist)*(entry),hipMemcpyHostToDevice,STREAM[stream_id]));
  HANDLE_ERROR(hipMemcpyAsync(D_SEARCH_NODE[stream_id],&(chunk->node_cube),sizeof(search_node_cube),hipMemcpyHostToDevice,STREAM[stream_id]));
  HANDLE_ERROR(hipMemcpyAsync(D_PHASE2_ENTRY[stream_id],&entry,sizeof(int),hipMemcpyHostToDevice,STREAM[stream_id]));
  HANDLE_ERROR(hipMemcpyAsync(D_RESULT[stream_id],&result_init,sizeof(int),hipMemcpyHostToDevice,STREAM[stream_id]));

  search_tree_phase2_cuda_karnel<<<entry/BLOCK_DIM,BLOCK_DIM,0,STREAM[stream_id]>>>(
    D_SEARCH_NODE[stream_id],
    D_MOVE_HIST[stream_id],
    D_PHASE2_ENTRY[stream_id],
    D_RESULT[stream_id],
    CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_POSITION[device],
    CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_ORIENTATION[device],
    CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_4_POSITION[device],
    CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_12_FLIP[device],
    CUDA_SOLVER_DISTANCE_TABLE[device],
    CUDA_SOLVER_CORNER_DISTANCE_TABLE[device],
    CUDA_SOLVER_CORNER_ORIENTATION_SYM_to_SYM0[device]
  );
}

int32_t get_search_result(int8_t *result, int32_t stream_id){
  int32_t result__[20];
  int32_t i;

  //printf("get_search_result stream = %d\n",stream_id);

  hipSetDevice(stream_id/2);

  HANDLE_ERROR(hipMemcpyAsync(result__,D_RESULT[stream_id],sizeof(int32_t)*20,hipMemcpyDeviceToHost, STREAM[stream_id]));
  hipStreamSynchronize(STREAM[stream_id]);
  if(result__[0] >= 0){
    for(i = 0;i < 20; i++){
      result[i] = result__[i];
    }
    return 1;
  }
  return 0;
}


int32_t get_n_exec_stream(){
  return N_STREAM;
}

void get_stream_state(stream_state_ptr state){
  int32_t i;
  for(i = 0; i <N_STREAM ; i++){
    if (i % 2 == 0){
      hipSetDevice(i/2);
    }
    if(hipStreamQuery(STREAM[i]) == hipErrorNotReady){
      state->stream_state_array[i] = STREAM_STATE_BUSY;
    }else{
      state->stream_state_array[i] = STREAM_STATE_READY;
      //printf("ready %d  %d\n", i, state->stream_state_array[i]);
    }
    //printf("aa %d  %d\n", i, state->stream_state_array[i]);
  }
}

void sync_all_stream(){
  int32_t i;
  //printf("sync_all\n");
  for(i = 0; i <N_STREAM ; i++){
    if (i % 2 == 0){
      hipSetDevice(i/2);
    }
    hipStreamSynchronize(STREAM[i]);
  }
}

void init_device_table(
  uint16_t * search_node_level_mov_tab_corner_8_ori,
  uint16_t * coordinate_level_mov_tab_corner_8_pos,
  uint16_t * search_node_level_mov_tab_edge_12_flip,
  uint16_t * search_node_level_mov_tab_edge_4_pos,
  uint16_t * corner_ori_sym_tab,
  uint16_t * edge_flip_sym_tab,
  uint8_t * distance_table,
  uint8_t * corner_distance_table,
  uint8_t * mov_trs_lr,
  uint8_t * mov_trs_fb
){
  static const uint8_t invalid_move[7] = { 0x21, 0x0A, 0x14, 0x08, 0x10, 0x20, 0x00 };
  int32_t i;


  for(i = 0;i < N_STREAM ; i++){
    if (i % 2 == 0){
      hipSetDevice(i/2);
    }
    HANDLE_ERROR(hipMalloc((void**)&D_MOVE_HIST[i],sizeof(move_hist)*((PHASE_2_SEARCH_CHUNK_SIZE_GPU+BLOCK_DIM-1)/BLOCK_DIM*BLOCK_DIM)));
    HANDLE_ERROR(hipMalloc((void**)&D_SEARCH_NODE[i],sizeof(search_node_cube)));
    HANDLE_ERROR(hipMalloc((void**)&D_PHASE2_ENTRY[i],sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&D_RESULT[i],sizeof(int32_t)*(20)));
  }

  for(i = 0;i < N_STREAM/2 ; i++){
    hipSetDevice(i);

    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_ORIENTATION[i],sizeof(uint16_t)*(N_CORNER_ORI*N_MOVES)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_POSITION[i]   ,sizeof(uint16_t)*(N_CORNER_POS*N_MOVES)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_12_FLIP[i]        ,sizeof(uint16_t)*(N_EDGE_FLIP*N_MOVES)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_4_POSITION[i]     ,sizeof(uint16_t)*(N_EDGE_4_POS*N_MOVES)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_CORNER_ORIENTATION_SYM_to_SYM0[i],sizeof(uint16_t)*(N_CORNER_ORI*N_SYM)));
    //HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0         ,sizeof(uint16_t)*(N_EDGE_FLIP*N_SYM)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_DISTANCE_TABLE[i]              ,sizeof(uint8_t)*(DISTANCE_TABLE_SIZE)));
    HANDLE_ERROR(hipMalloc((void**)&CUDA_SOLVER_CORNER_DISTANCE_TABLE[i]              ,sizeof(uint8_t)*(CORNER_DISTANCE_TABLE_SIZE)));

    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_ORIENTATION[i], search_node_level_mov_tab_corner_8_ori,sizeof(uint16_t)*N_CORNER_ORI*N_MOVES,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_CORNER_8_POSITION[i]   , coordinate_level_mov_tab_corner_8_pos,sizeof(uint16_t)*N_CORNER_POS*N_MOVES,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_12_FLIP[i]        , search_node_level_mov_tab_edge_12_flip,sizeof(uint16_t)*N_EDGE_FLIP *N_MOVES,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_SEARCH_NODE_LEVEL_MOV_TAB_EDGE_4_POSITION[i]     , search_node_level_mov_tab_edge_4_pos  ,sizeof(uint16_t)*N_EDGE_4_POS*N_MOVES,hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_CORNER_ORIENTATION_SYM_to_SYM0[i], corner_ori_sym_tab  ,sizeof(uint16_t)*N_CORNER_ORI*N_SYM,hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0         , edge_flip_sym_tab   ,sizeof(uint16_t)*N_EDGE_FLIP *N_SYM,hipMemcpyHostToDevice));


    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_DISTANCE_TABLE[i], distance_table   ,sizeof(uint8_t)*DISTANCE_TABLE_SIZE,hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(CUDA_SOLVER_CORNER_DISTANCE_TABLE[i], corner_distance_table   ,sizeof(uint8_t)*CORNER_DISTANCE_TABLE_SIZE,hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_SOLVER_MOV_TRS_TAB_LR),mov_trs_lr, sizeof(uint8_t) *N_MOVES));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_SOLVER_MOV_TRS_TAB_FB),mov_trs_fb, sizeof(uint8_t) *N_MOVES));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(D_INVALID_MOVE ),invalid_move, sizeof(uint8_t) *7));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_SOLVER_EDGE_FLIP_SYM_to_SYM0 ), edge_flip_sym_tab,sizeof(uint16_t)*N_EDGE_FLIP *N_SYM));

  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(search_tree_phase2_cuda_karnel),hipFuncCachePreferL1);

  }

  for(i = 0;i < N_STREAM ; i++){
    if (i % 2 == 0){
      hipSetDevice(i/2);
    }
    HANDLE_ERROR(hipStreamCreate(&STREAM[i]));
  }
}

#endif
